#include "hip/hip_runtime.h"
#include "stdlib.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <chrono>


    
// #define BLOCK_SIZE  4
#define FILTER_WINDOW_SIZE 9
// #define WINDOW_SIZE (BLOCK_SIZE*BLOCK_SIZE*FILTER_WINDOW_SIZE*sizeof(float))

enum XORY{
    X_SOBEL = 0,
    Y_SOBEL = 1
};

typedef unsigned int uint;
void createGoldenStandard( float *origData, float *standData, unsigned int width, unsigned int height, int xory);
float compareToStandard( float *standData, float *testData, uint width, uint height);

const char *statsFileName = "out_stats.txt";
const char *goldenStandardX = "data/sobel_gs_x.pgm";
const char *goldenStandardY = "data/sobel_gs_y.pgm";
const int sobel_x[3][3] = {{-1, 0, 1}, {-2, 0 , 2}, {-1, 0, 1}};
const int sobel_y[3][3] = {{-1, -2, 1}, {0, 0, 0}, {1, 2, 1}};

// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;


// Kernel function to perform blurring
__global__ void sobelKernel(float *outputData, int width, int height, int xory){
    extern __shared__ float window[];
    const int sobel_x[3][3] = {{-1, 0, 1}, {-2, 0 , 2}, {-1, 0, 1}};
    const int sobel_y[3][3] = {{-1, -2, 1}, {0, 0, 0}, {1, 2, 1}};

    // calculate texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int pixelRadius = 1;

    if(x < pixelRadius || y < pixelRadius || x >= (width-pixelRadius) || y >= (height-pixelRadius)){
        // Do nothing
        outputData[(y * width) + x] = tex2D(tex, x, y);
    }
    else{
        // Calculate bounds for processing
        uint x_start    = x - pixelRadius;
        uint x_end      = x + pixelRadius;
        uint y_start    = y - pixelRadius;
        uint y_end      = y + pixelRadius;

        float sum = 0;

        // Fill array with values
        for(int i = x_start, ii=0 ; i <= x_end; i++, ii++){
            for(int j =  y_start, jj=0 ; j <= y_end ; j++, jj++){
                if(xory == X_SOBEL){
                    sum += tex2D(tex, i, j) * sobel_x[ii][jj];                    
                }
                else if(xory == Y_SOBEL){
                    sum += tex2D(tex, i, j) * sobel_y[ii][jj];                    
                }
            }
        }
        outputData[(y * width) + x] = sum;
    }
}


int main(int argc, char **argv){
    // Check args
    if(argc < 5){
        #if DEBUG_MESSAGES_ON
        std::cout << "\n\nIncorrect number of args.  Should be \n(1)input file\n(2)X output file\n(3)Y output file\n(4)blockSize X\n(5)blockSize Y" << std::endl;
        #endif
        return 0;
    }
    // Get input and output files
    char *inputfile = argv[1];
    if(!inputfile){
        return 0;
    }
    char *xoutputfile = argv[2];
    if(!xoutputfile){
        return 0;
    }
    char *youtputfile = argv[3];
    if(!youtputfile){
        return 0;
    }
    int blockSizeX = atoi(argv[4]);
    if(blockSizeX <= 0){
        return 0;
    }
    int blockSizeY = atoi(argv[5]);
    if(blockSizeY <= 0){
        return 0;
    }

    #if TEST_MODE
    std::cout << "\n\n\nTesting " << blockSizeX << " x " << blockSizeY << " blocks" << std::endl;
    #endif
    // Load PGM onto device
    int devID = findCudaDevice(argc, (const char **) argv);

    #if TEST_MODE
    FILE *statsFile = fopen(statsFileName, "a");
    if(!statsFile){
        std::cout << "Couldn't open stats file" << std::endl;
    }
    #endif

    /***************
    LOAD INPUT FILE
    ****************/
    // Start timing for load time
    std::chrono::time_point <std::chrono::system_clock> start, end;

    float *origData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(inputfile, argv[0]);

    if (imagePath == NULL){
        #if DEBUG_MESSAGES_ON
        std::cout << "Unable to source image file:"<< inputfile << " %s\n" << std::endl;
        #endif
        exit(EXIT_FAILURE);
    }
    start = std::chrono::system_clock::now();
    sdkLoadPGM(imagePath, &origData, &width, &height);
    end = std::chrono::system_clock::now();

    // Print to stats file
    #if TEST_MODE
    if(statsFile){
        std::chrono::duration<double> timeElapsed = end-start;
        fprintf(statsFile, "LoadTimeStats: %f\n", timeElapsed);
    }
    #endif

    int size = width * height * sizeof(float);
    #if DEBUG_MESSAGES_ON
    std::cout << "Loaded " << inputfile << ", " << width << " x "<< height << " pixels with size " << (uint)size << std::endl;
    #endif


    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *inArray;
    checkCudaErrors(hipMallocArray(&inArray,
                                    &channelDesc,
                                    width,
                                    height));
    // Start timer
    start = std::chrono::system_clock::now();
    checkCudaErrors(hipMemcpyToArray(inArray,
                                      0,
                                      0,
                                      origData,
                                      size,
                                      hipMemcpyHostToDevice));

    #if DEBUG_MESSAGES_ON
    std::cout << "\nLoaded " << inputfile << " onto device." << std::endl;
    #endif


    /************
    SET UP TEXTURE AND GRID INFORMATION
    *************/
    // Set texture parameters
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;    // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, inArray, channelDesc));

    // Set up grid
    dim3 dimBlock(blockSizeX, blockSizeY, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
    // int window_size = BLOCK_SIZE * BLOCK_SIZE * filterSize * filterSize * sizeof(float);


    #if DEBUG_MESSAGES_ON
    std::cout << "\nBlocks and grid set up.\nBlock is " << dimBlock.x << " x " << dimBlock.y << \
        "\nGrid is " << dimGrid.x << " x " << dimGrid.y << std::endl;
    #endif


    /***************
    CALL KERNEL TO PROCESS FILE
    ****************/
    // Allocate device memory for result
    float *outData = NULL;
    checkCudaErrors(hipMalloc((void **) &outData, size));
    sobelKernel<<<dimGrid, dimBlock>>>(outData, width, height, X_SOBEL);
    getLastCudaError("Kernel execution failed");


    /***************
    SAVE X OUTPUT TO FILE
    ****************/
    // Allocate mem for the result on host side
    float *hOutputData = (float *) malloc(size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(hOutputData,
                               (const void*)outData,
                               size,
                               hipMemcpyDeviceToHost));
    end = std::chrono::system_clock::now();
    // Write to stats file
    #if TEST_MODE
    if(statsFile){
        std::chrono::duration<double> timeElapsed = end-start;
        fprintf(statsFile, "ComputeTimeStats: %s %d  %f\n", "x", blockSizeX*blockSizeY, timeElapsed);
    }
    #endif


    // Write to file
    char *outimagePath = sdkFindFilePath(xoutputfile, argv[0]);
    if (outimagePath == NULL){
        // #if DEBUG_MESSAGES_ON
        std::cout << "Unable to source image file:"<< xoutputfile << "\n" << std::endl;
        // #endif
        exit(EXIT_FAILURE);
    }
    sdkSavePGM(outimagePath, hOutputData, width, height);
    // #if DEBUG_MESSAGES_ON
    std::cout << "Wrote to " << xoutputfile << "." << std::endl;
    // #endif


    /****************
    Do Y sobel filter now
    *****************/
    // Allocate device memory for result
    float *yOutData = NULL;
    checkCudaErrors(hipMalloc((void **) &yOutData, size));
    sobelKernel<<<dimGrid, dimBlock>>>(yOutData, width, height, Y_SOBEL);
    getLastCudaError("Kernel execution failed");


    /***************
    SAVE Y OUTPUT TO FILE
    ****************/
    // Allocate mem for the result on host side
    float *hyOutputData = (float *) malloc(size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(hyOutputData,
                               (const void*)yOutData,
                               size,
                               hipMemcpyDeviceToHost));
    end = std::chrono::system_clock::now();
    // Write to stats file
    #if TEST_MODE
    if(statsFile){
        std::chrono::duration<double> timeElapsed = end-start;
        fprintf(statsFile, "ComputeTimeStats: %s %d  %f\n", "y", blockSizeX * blockSizeY, timeElapsed);
    }
    #endif


    // Write to file
    char *youtimagePath = sdkFindFilePath(youtputfile, argv[0]);
    if (youtimagePath == NULL){
        // #if DEBUG_MESSAGES_ON
        std::cout << "Unable to source image file:"<< xoutputfile << "\n" << std::endl;
        // #endif
        exit(EXIT_FAILURE);
    }
    sdkSavePGM(youtimagePath, hyOutputData, width, height);
    // #if DEBUG_MESSAGES_ON
    std::cout << "Wrote to " << youtputfile << "." << std::endl;
    // #endif

    /*************
    CREATE STANDARD FILE TO TEST CUDA SOLUTION ON HOST
    **************/
    #if TEST_MODE
    start = std::chrono::system_clock::now();
    // Allocate mem for the standards
    float *gsXData = (float *) malloc(size);
    float *gsYData = (float *) malloc(size);
    createGoldenStandard(origData, gsXData, width, height, X_SOBEL);
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> timeElapsed = end-start;
    // Print timing to file
    if(statsFile){
        fprintf(statsFile, "GSTimingStats: %f\n", timeElapsed);
    }


    start = std::chrono::system_clock::now();
    createGoldenStandard(origData, gsYData, width, height, Y_SOBEL);
    end = std::chrono::system_clock::now();
    timeElapsed = end-start;
    // Print timing to file
    if(statsFile){
        fprintf(statsFile, "GSTimingStats: %f\n", timeElapsed);
    }    

    // Compare X output to standard, get percentage correct back
    float percentage = compareToStandard(gsXData, hOutputData, width, height);
    std::cout << "Percentage correct: " << percentage << "%" << std::endl;
    if(statsFile){
        fprintf(statsFile, "AccuracyStats: %f\n", percentage);
    }else{
        std::cout << "Couldn't write to stats file" <<std::endl;
    }

    percentage = compareToStandard(gsYData, hyOutputData, width, height);
    std::cout << "Percentage correct: " << percentage << "%" << std::endl;
    if(statsFile){
        fprintf(statsFile, "AccuracyStats: %f\n", percentage);
    }
    // Print to file if specified
    char *stdXOutImagePath = sdkFindFilePath(goldenStandardX, argv[0]);
    if (stdXOutImagePath == NULL){
        // #if DEBUG_MESSAGES_ON
        std::cout << "Unable to source image file:"<< goldenStandardX << "\n" << std::endl;
        // #endif
        exit(EXIT_FAILURE);
    }
    sdkSavePGM(stdXOutImagePath, gsXData, width, height);
    
    char *stdYOutImagePath = sdkFindFilePath(goldenStandardY, argv[0]);
    if (stdYOutImagePath == NULL){
        // #if DEBUG_MESSAGES_ON
        std::cout << "Unable to source image file:"<< goldenStandardY << "\n" << std::endl;
        // #endif
        exit(EXIT_FAILURE);
    }
    sdkSavePGM(stdYOutImagePath, gsYData, width, height);
    #endif

    return 0;
}


void createGoldenStandard( float *origData, float *standData, unsigned int width, unsigned int height, int xory){
    if(origData == NULL || standData == NULL){
        // #if DEBUG_MESSAGES_ON
        std::cout << "Data is null" << std::endl;
        // #endif
        return;
    }

    uint pixelRadius = 1;
         // arraySize = FILTER_WINDOW_SIZE;
        // halfArraySize = arraySize/2 + 1;


    for(int y = 0 ; y <= height ; y++){
        for(int x = 0 ; x <= width ; x++){
            if(x < pixelRadius || x > width-pixelRadius || y <= pixelRadius || y >= height-pixelRadius){
                standData[(y*width) + x] = origData[(y*width) + x];
            }
            else{
                // At 1 pixel currently.  Iterate through its neighbors and find median.
                uint p_x_start = x - pixelRadius,
                     p_x_end   = x + pixelRadius,
                     p_y_start = y - pixelRadius,
                     p_y_end   = y + pixelRadius;

                float sum = 0;
                // Add neighbors to neighbors array
                for(int i = 0, yy = p_y_start ; yy <= p_y_end ; i++, yy++){
                    for(int j = 0, xx = p_x_start ; xx <= p_x_end ; j++, xx++){
                        if(xory == X_SOBEL){
                            sum += origData[(yy * width) + xx] * sobel_x[i][j];
                        }
                        else{
                            sum += origData[(yy * width) + xx] * sobel_y[i][j];
                        }
                    }
                }
                standData[(y*width) + x] = sum;
            }
        }
    }
    return;
}

float compareToStandard(float *standData, float *testData, uint width, uint height){
    if(standData == NULL || testData == NULL){
        return 0;
    }

    uint count = 0, numCorrect = 0;
    for(int y = 0 ; y < height ; y++){
        for(int x = 0 ; x < width ; x++){
            if(standData[(y*width) + x] == testData[(y*width) + x]){
                numCorrect++;
            }
            else{
                // std::cout << standData[(y*width) + x] << " vs " << testData[(y*width) + x] << std::endl;
            }
            count++;
        }
    }
    return (float)100.0*(float)((float)numCorrect/(float)count);
}









